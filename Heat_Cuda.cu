#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

// Simple define to index into a 1D array from 2D space
#define I2D(num, c, r) ((r)*(num)+(c))

/*
  Cuda permette di sfruttare la GPU per lanciare diversi threads su una medesima operazione, ma su dati differenti.
  Per far questo bisogna definire il numero di blocchi e il numero di threads per ogni blocco.
  I threads sono raggrupati in batch chiamati warp e sono 32.

  Per N elementi da calcolare nella CPU, necessito di lanciare N threads.

  Solitamente bisogna gestire la copia e lo spostamento dei dati da una memoria all'altra (CPU - GPU).
  Se usassimo CudaMallocManaged, tutto questo viene gestito in maniera automatica.

  Gerarchia:
  - I thread sono raggruppati in blocchi.
  - I blocchi possono essere  1D, 2D o 3D.
  - I blocchi sono  organizzati in una griglia, che può essere 1D, 2D o 3D.

  Ogni thread e blocco ha un identificatore unico:
  - I thread all'interno di un blocco sono indicizzati usando la variabile threadIdx, che contiene le componenti x, y e z per i blocchi multidimensionali.
  - I blocchi all'interno di una griglia sono indicizzati usando la variabile blockIdx, con componenti x, y e z per le griglie multidimensionali.

  La dimensione di ciascuna dimensione è accessibile tramite:
  - blockDim, che fornisce le dimensioni di un blocco in termini di thread.
  - gridDim, che fornisce le dimensioni della griglia in termini di blocchi.

    Griglia (2D) [composta da blocchi]

    | Block (0,0) | Block (1,0) | Block (2,0) |
    |-------------|-------------|-------------|
    | Block (0,1) | Block (1,1) | Block (2,1) |
    |-------------|-------------|-------------|
    | Block (0,2) | Block (1,2) | Block (2,2) |

    Blocco (2,1) [composta da Thread]
    |-Thread (0,0) Thread (1,0) Thread (2,0) Thread (3,0)|
    |-Thread (0,1) Thread (1,1) Thread (2,1) Thread (3,1)|
    |-Thread (0,2) Thread (1,2) Thread (2,2) Thread (3,2)|
    |-Thread (0,3) Thread (1,3) Thread (2,3) Thread (3,3)|

*/

__global__ void step_kernel_mod(int ni, int nj, float fact, float* temp_in, float* temp_out)
{
  int i00, im10, ip10, i0m1, i0p1;
  float d2tdx2, d2tdy2;

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < (nj-2)*(ni-2)){

      int i = idx % (ni-2) + 1;
      int j = idx / (ni-2) + 1;

      if (i < ni-1 && j < nj-1){
      i00 = I2D(ni, i, j);
      im10 = I2D(ni, i-1, j);
      ip10 = I2D(ni, i+1, j);
      i0m1 = I2D(ni, i, j-1);
      i0p1 = I2D(ni, i, j+1);

      // evaluate derivatives
      d2tdx2 = temp_in[im10]-2*temp_in[i00]+temp_in[ip10];
      d2tdy2 = temp_in[i0m1]-2*temp_in[i00]+temp_in[i0p1];

      // update temperatures
      temp_out[i00] = temp_in[i00]+fact*(d2tdx2 + d2tdy2);
      }
    }
}

void step_kernel_ref(int ni, int nj, float fact, float* temp_in, float* temp_out)
{
  int i00, im10, ip10, i0m1, i0p1;
  float d2tdx2, d2tdy2;


  // loop over all points in domain (except boundary)
  for ( int j=1; j < nj-1; j++ ) {
    for ( int i=1; i < ni-1; i++ ) {
      // find indices into linear memory
      // for central point and neighbours
      i00 = I2D(ni, i, j);
      im10 = I2D(ni, i-1, j);
      ip10 = I2D(ni, i+1, j);
      i0m1 = I2D(ni, i, j-1);
      i0p1 = I2D(ni, i, j+1);

      // evaluate derivatives
      d2tdx2 = temp_in[im10]-2*temp_in[i00]+temp_in[ip10];
      d2tdy2 = temp_in[i0m1]-2*temp_in[i00]+temp_in[i0p1];

      // update temperatures
      temp_out[i00] = temp_in[i00]+fact*(d2tdx2 + d2tdy2);
    }
  }
}

int main()
{
  int istep;
  int nstep = 200; // number of time steps

  // Specify our 2D dimensions (MODIFICARE CON 10 000 X 10 000 E CON 30 000 X 30 000)
  const int ni = 1000;
  const int nj = 1000;
  float tfac = 8.418e-5; // thermal diffusivity of silver

  float *temp1_ref, *temp2_ref, *temp1, *temp2, *temp_tmp;

  const int size = ni * nj * sizeof(float);

  temp1_ref = (float*)malloc(size);
  temp2_ref = (float*)malloc(size);

//  GESTIONE DELLE VARIABILI CONDIVISE CPU-GPU

  hipMallocManaged(&temp1, size);
  hipMallocManaged(&temp2, size);
  
/*  
    DEFINITION OF THE BLOCKS AND THREADS
    Threads must be multiple of 32 (Cuda GuideLine) 
 */

  // Total number of internal points in the domain
  int numPoints = (ni-2) * (nj-2); 
  dim3 threads(64);  
  dim3 dimblock((numPoints + threads.x - 1) / threads.x);  // Dimension of the block proportional to the number of points

  // print some info
  printf("threads: %u \n", threads.x);
  printf("block: %u \n", dimblock.x);
  printf("total num of threads: %u\n", threads.x*dimblock.x);
  printf("total num of pixels: %d\n", ni*nj);
  printf("total num of internal pixels: %d\n", (ni-2) * (nj-2));
 
  //Identify some events to measure the time
  hipEvent_t start_ref, stop_ref; 
  hipEvent_t start_mod, stop_mod;
  hipEventCreate(&start_ref);
  hipEventCreate(&stop_ref);
  hipEventCreate(&start_mod);
  hipEventCreate(&stop_mod);

  // Initialize with random data
  for( int i = 0; i < ni*nj; ++i) {
    temp1_ref[i] = temp2_ref[i] = temp1[i] = temp2[i] = (float)rand()/(float)(RAND_MAX/100.0f);
  }

  hipEventRecord(start_ref);

  // Execute the CPU-only reference version
  for (istep=0; istep < nstep; istep++) {
    step_kernel_ref(ni, nj, tfac, temp1_ref, temp2_ref);

    temp_tmp = temp1_ref; // swap the temperature pointers
    temp1_ref = temp2_ref;
    temp2_ref= temp_tmp;
  }

  hipEventRecord(stop_ref);  
  hipEventSynchronize(stop_ref);
  float elapsed_ref = 0;
  // Compute the elapsed time
  hipEventElapsedTime(&elapsed_ref, start_ref, stop_ref);
  // Print the elapsed time
  printf("Elapsed ref time: %f microseconds\n", elapsed_ref) ; 

  hipEventRecord(start_mod);

  // Execute the modified version using same data
  for (istep=0; istep < nstep; istep++) {

    // original problem without blocks and threads
    //step_kernel_mod(ni, nj, tfac, temp1, temp2);
    
    // i have to specify the number of threads e dimblock
    step_kernel_mod<<< dimblock , threads >>>(ni, nj, tfac, temp1, temp2);
    hipDeviceSynchronize();

    // swap the temperature pointers
    temp_tmp = temp1;
    temp1 = temp2;
    temp2= temp_tmp;
  }

  hipEventRecord(stop_mod); 
  hipEventSynchronize(stop_mod);
  float elapsed_mod = 0;
  hipEventElapsedTime(&elapsed_mod, start_mod, stop_mod);
  printf("Elapsed mod time: %f microseconds\n", elapsed_mod) ; 

  // Check for errors (all CUDA API calls return an error code)
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

  float maxError = 0;
  // Output should always be stored in the temp1 and temp1_ref at this point
  for( int i = 0; i < ni*nj; ++i ) {
    if (abs(temp1[i]-temp1_ref[i]) > maxError) { maxError = abs(temp1[i]-temp1_ref[i]); }
  }

  // Check and see if our maxError is greater than an error bound
  if (maxError > 0.0005f)
    printf("Problem! The Max Error of %.5f is NOT within acceptable bounds.\n", maxError);
  else
    printf("The Max Error of %.5f is within acceptable bounds.\n", maxError);

  
  
  // se omettimo il calcolo dell'errore possiamo non deallocare temp1_ref e temp2_ref
  hipFree(temp1);
  hipFree(temp2);
  
  free( temp1_ref );
  free( temp2_ref );

  return 0;
}
